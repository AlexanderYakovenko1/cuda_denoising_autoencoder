#include <iostream>
#include <vector>

#include "include/cnpy.h"
#include "utils.h"
#include "layers.cuh"

constexpr int BLOCK_SIZE = 32;
constexpr int MAX_THREADS = 1024;

std::pair<int64_t, int64_t> DenoiseMNIST(const float* in, float* out, int height, int width, const std::string& weights_dir="../model_weights") {
    std::chrono::steady_clock::time_point all_begin = std::chrono::steady_clock::now();

    cnpy::NpyArray conv1_weight = cnpy::npy_load(weights_dir + "/conv1.weight.npy");
    cnpy::NpyArray conv1_bias = cnpy::npy_load(weights_dir + "/conv1.bias.npy");
    cnpy::NpyArray conv2_weight = cnpy::npy_load(weights_dir + "/conv2.weight.npy");
    cnpy::NpyArray conv2_bias = cnpy::npy_load(weights_dir + "/conv2.bias.npy");

    cnpy::NpyArray upconv1_weight = cnpy::npy_load(weights_dir + "/upconv1.weight.npy");
    cnpy::NpyArray upconv1_bias = cnpy::npy_load(weights_dir + "/upconv1.bias.npy");
    cnpy::NpyArray upconv2_weight = cnpy::npy_load(weights_dir + "/upconv2.weight.npy");
    cnpy::NpyArray upconv2_bias = cnpy::npy_load(weights_dir + "/upconv2.bias.npy");
    cnpy::NpyArray conv3_weight = cnpy::npy_load(weights_dir + "/conv3.weight.npy");
    cnpy::NpyArray conv3_bias = cnpy::npy_load(weights_dir + "/conv3.bias.npy");

    float* cuda_conv1_weight = nullptr;
    float* cuda_conv1_bias = nullptr;
    float* cuda_conv2_weight = nullptr;
    float* cuda_conv2_bias = nullptr;

    float* cuda_upconv1_weight = nullptr;
    float* cuda_upconv1_bias = nullptr;
    float* cuda_upconv2_weight = nullptr;
    float* cuda_upconv2_bias = nullptr;
    float* cuda_conv3_weight = nullptr;
    float* cuda_conv3_bias = nullptr;

    CHECK_CUDA_ERRS( hipMalloc(&cuda_conv1_weight, conv1_weight.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_conv1_weight, conv1_weight.data<float>(), conv1_weight.num_vals * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_conv1_bias, conv1_bias.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_conv1_bias, conv1_bias.data<float>(), conv1_bias.num_vals * sizeof(float), hipMemcpyHostToDevice) );

    CHECK_CUDA_ERRS( hipMalloc(&cuda_conv2_weight, conv2_weight.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_conv2_weight, conv2_weight.data<float>(), conv2_weight.num_vals * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_conv2_bias, conv2_bias.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_conv2_bias, conv2_bias.data<float>(), conv2_bias.num_vals * sizeof(float), hipMemcpyHostToDevice) );

    CHECK_CUDA_ERRS( hipMalloc(&cuda_upconv1_weight, upconv1_weight.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_upconv1_weight, upconv1_weight.data<float>(), upconv1_weight.num_vals * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_upconv1_bias, upconv1_bias.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_upconv1_bias, upconv1_bias.data<float>(), upconv1_bias.num_vals * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_upconv2_weight, upconv2_weight.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_upconv2_weight, upconv2_weight.data<float>(), upconv2_weight.num_vals * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_upconv2_bias, upconv2_bias.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_upconv2_bias, upconv2_bias.data<float>(), upconv2_bias.num_vals * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_conv3_weight, conv3_weight.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_conv3_weight, conv3_weight.data<float>(), conv3_weight.num_vals * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_conv3_bias, conv3_bias.num_vals * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_conv3_bias, conv3_bias.data<float>(), conv3_bias.num_vals * sizeof(float), hipMemcpyHostToDevice) );

    int half_width = width >> 1;
    int half_height = height >> 1;
    int quarter_width = width >> 2;
    int quarter_height = height >> 2;
    int full_size = width * height;
    int half_size = half_width * half_height;
    int quarter_size = quarter_width * quarter_height;

    float* cuda_fullres1 = nullptr;
    float* cuda_fullres32_1 = nullptr;
    float* cuda_fullres32_2 = nullptr;
    float* cuda_halfres32_1 = nullptr;
    float* cuda_halfres32_2 = nullptr;
    float* cuda_quarterres32 = nullptr;

    CHECK_CUDA_ERRS( hipMalloc(&cuda_fullres1, full_size * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMemcpy(cuda_fullres1, in, full_size * sizeof(float), hipMemcpyHostToDevice) );

    CHECK_CUDA_ERRS( hipMalloc(&cuda_fullres32_1, 32 * full_size * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_fullres32_2, 32 * full_size * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_halfres32_1, 32 * half_size * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_halfres32_2, 32 * half_size * sizeof(float)) );
    CHECK_CUDA_ERRS( hipMalloc(&cuda_quarterres32, 32 * quarter_size * sizeof(float)) );

//
    // todo: remore, this is for debug only
//    float* cpu_out = static_cast<float *>(malloc(32 * full_size * sizeof(float)));
//

    dim3 numBlocksFull(width / BLOCK_SIZE + 1, height / BLOCK_SIZE + 1);
    dim3 numBlocksHalf(half_width / BLOCK_SIZE + 1, half_height / BLOCK_SIZE + 1);
    dim3 numBlocksQuarter(quarter_width / BLOCK_SIZE + 1, quarter_height / BLOCK_SIZE + 1);
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    Conv2D_3x3<<<numBlocksFull, threadsPerBlock>>>(
            cuda_fullres1, cuda_fullres32_1, cuda_conv1_weight, cuda_conv1_bias, 1, 32, height, width);
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error1: %s\n", hipGetErrorString(error)); exit(-1);
    }

//
//    CHECK_CUDA_ERRS( hipMemcpy(cpu_out, cuda_fullres32_1, 32 * full_size * sizeof(float), hipMemcpyDeviceToHost) );
//    cnpy::npy_save("../0_conv1.npy", cpu_out, {32, 256, 256});
//

    ReLU<<<32 * full_size / MAX_THREADS + 1, MAX_THREADS>>>(cuda_fullres32_1, cuda_fullres32_1, 32, height, width);

    MaxPool2D<<<numBlocksHalf, threadsPerBlock>>>(cuda_fullres32_1, cuda_halfres32_1, 32, height, width, 2);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error2: %s\n", hipGetErrorString(error)); exit(-1);
    }
//
//    CHECK_CUDA_ERRS( hipMemcpy(cpu_out, cuda_halfres32_1, 32 * half_size * sizeof(float), hipMemcpyDeviceToHost) );
//    cnpy::npy_save("../1_maxpool1.npy", cpu_out, {32, 128, 128});
//

    Conv2D_3x3<<<numBlocksHalf, threadsPerBlock>>>(
            cuda_halfres32_1, cuda_halfres32_2, cuda_conv2_weight, cuda_conv2_bias, 32, 32, half_height, half_width);
//
//    CHECK_CUDA_ERRS( hipMemcpy(cpu_out, cuda_halfres32_2, 32 * half_size * sizeof(float), hipMemcpyDeviceToHost) );
//    cnpy::npy_save("../2_conv2.npy", cpu_out, {32, 128, 128});
//
    ReLU<<<32 * half_size / MAX_THREADS + 1, MAX_THREADS>>>(cuda_halfres32_2, cuda_halfres32_2, 32, half_height, half_width);

    MaxPool2D<<<numBlocksQuarter, threadsPerBlock>>>(cuda_halfres32_2, cuda_quarterres32, 32, half_height, half_width, 2);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error3: %s\n", hipGetErrorString(error)); exit(-1);
    }
//
//    CHECK_CUDA_ERRS( hipMemcpy(cpu_out, cuda_quarterres32, 32 * quarter_size * sizeof(float), hipMemcpyDeviceToHost) );
//    cnpy::npy_save("../3_maxpool2.npy", cpu_out, {32, 64, 64});
//

    TranposedConv2D_3x3_2<<<numBlocksHalf, threadsPerBlock>>>(
            cuda_quarterres32, cuda_halfres32_1, cuda_halfres32_2, cuda_upconv1_weight, cuda_upconv1_bias, 32, 32, quarter_height, quarter_width);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error4: %s\n", hipGetErrorString(error)); exit(-1);
    }
//
//    CHECK_CUDA_ERRS( hipMemcpy(cpu_out, cuda_halfres32_1, 32 * half_size * sizeof(float), hipMemcpyDeviceToHost) );
//    cnpy::npy_save("../4_upconv1.npy", cpu_out, {32, 128, 128});
//
    ReLU<<<32 * half_size / MAX_THREADS + 1, MAX_THREADS>>>(cuda_halfres32_1, cuda_halfres32_1, 32, half_height, half_width);

    TranposedConv2D_3x3_2<<<numBlocksFull, threadsPerBlock>>>(
            cuda_halfres32_1, cuda_fullres32_1, cuda_fullres32_2, cuda_upconv2_weight, cuda_upconv2_bias, 32, 32, half_height, half_width);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error5: %s\n", hipGetErrorString(error)); exit(-1);
    }
//
//    CHECK_CUDA_ERRS( hipMemcpy(cpu_out, cuda_fullres32_1, 32 * full_size * sizeof(float), hipMemcpyDeviceToHost) );
//    cnpy::npy_save("../5_upconv2.npy", cpu_out, {32, 256, 256});
//
    ReLU<<<32 * full_size / MAX_THREADS + 1, MAX_THREADS>>>(cuda_fullres32_1, cuda_fullres32_1, 32, height, width);

    Conv2D_3x3<<<numBlocksFull, threadsPerBlock>>>(
            cuda_fullres32_1, cuda_fullres1, cuda_conv3_weight, cuda_conv3_bias, 32, 1, height, width);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error6: %s\n", hipGetErrorString(error)); exit(-1);
    }
//
//    CHECK_CUDA_ERRS( hipMemcpy(cpu_out, cuda_fullres1, full_size * sizeof(float), hipMemcpyDeviceToHost) );
//    cnpy::npy_save("../6_conv3.npy", cpu_out, {1, 256, 256});
//
    Sigmoid<<<full_size / MAX_THREADS + 1, MAX_THREADS>>>(cuda_fullres1, cuda_fullres1, 1, height, width);
//
//    CHECK_CUDA_ERRS( hipMemcpy(cpu_out, cuda_fullres1, full_size * sizeof(float), hipMemcpyDeviceToHost) );
//    cnpy::npy_save("../out.npy", cpu_out, {1, 256, 256});
//
    hipDeviceSynchronize();

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

    CHECK_CUDA_ERRS( hipMemcpy(out, cuda_fullres1, full_size * sizeof(float), hipMemcpyDeviceToHost) );

    std::chrono::steady_clock::time_point all_end = std::chrono::steady_clock::now();

    return std::make_pair(std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count(),
                          std::chrono::duration_cast<std::chrono::microseconds>(all_end - all_begin).count());
}

int main(int argc, char **argv) {
    char* input_path;
    char* output_path;
    int num_runs;

    ParseArguments(argc, argv, &input_path, &output_path, &num_runs);
    printf("Denoising %s and saving the result to %s\n", input_path, output_path);

    int width, height, channels;
    uint8_t* image = stbi_load(input_path, &width, &height, &channels, STBI_grey);
    if (channels != 1) {
        printf("Only grayscale images supported. Provided image has %d channels.\n", channels);
    }
    float* in_float_image = AllocateArray<float>(width, height, channels);
    float* out_float_image = AllocateArray<float>(width, height, channels);
    UintToFloat(image, in_float_image, width, height, channels, 1.f/255.f);

    int64_t total_duration_with = 0;
    int64_t total_duration_without = 0;
    for (int n_run = 0; n_run < num_runs; ++n_run) {
        auto [without_allocs, with_allocs] = DenoiseMNIST(in_float_image, out_float_image, height, width);
        std::cout << "Run " << n_run << std::endl;
        std::cout << "Took " << without_allocs << "[µs] to complete (without allocation)" << std::endl;
        std::cout << "Took " << with_allocs << "[µs] to complete (with allocation)" << std::endl;
        total_duration_without += without_allocs;
        total_duration_with += with_allocs;
    }
    std::cout << "Mean runtime over " << num_runs << " runs: " << total_duration_without / num_runs << "[µs] (without allocation)" << std::endl;
    std::cout << "Mean runtime over " << num_runs << " runs: " << total_duration_with / num_runs << "[µs] (with allocation)" << std::endl;


    FloatToUint(out_float_image, image, width, height, channels, 255.f);

    SaveImage(image, output_path, width, height, channels);

    return 0;
}
